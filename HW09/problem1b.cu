#include<iostream>
#include<stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define RADIUS 3

int checkResults(int startElem, int endElem, float* cudaRes, float* res)
{
    int nDiffs=0;
    const float smallVal = 0.0001f;
    for(int i=startElem; i<endElem; i++)
        if(fabs(cudaRes[i]-res[i])>smallVal)
            nDiffs++;
    return nDiffs;
}

void initializeWeights(float* weights, int rad)
{
    // for now hardcoded for RADIUS=3
    weights[0] = 0.50f;
    weights[1] = 0.75f;
    weights[2] = 1.25f;
    weights[3] = 2.00f;
    weights[4] = 1.25f;
    weights[5] = 0.75f;
    weights[6] = 0.50f;
}
void initializeArray(FILE* fp,float* arr, int nElements)
{
    for( int i=0; i<nElements; i++){
                fscanf(fp,"%f",&arr[i]);
                if(getc(fp) == EOF) rewind(fp);
    }
}

void applyStencil1D_SEQ(int sIdx, int eIdx, const float *weights, float *in, float *out) {
  
  for (int i = sIdx; i < eIdx; i++) {   
    out[i] = 0;
    //loop over all elements in the stencil
    for (int j = -RADIUS; j <= RADIUS; j++) {
      out[i] += weights[j + RADIUS] * in[i + j]; 
    }
    out[i] = out[i] / (2 * RADIUS + 1);
  }
}

__global__ void applyStencil1D(int sIdx, int eIdx, const float *weights, float *in, float *out) {
    int i = sIdx + blockIdx.x*blockDim.x + threadIdx.x;
    __shared__ volatile float list[519];
    __shared__ volatile float sharedWeights[7];
    if(threadIdx.x < 7){
        //if the thread index is less that 7, load the shared weights
        sharedWeights[threadIdx.x] = weights[threadIdx.x];
		list[threadIdx.x] = in[i-3];
    }
    //each thread helps load one number
    list[threadIdx.x+6] = in[i+3];
    __syncthreads();
    
    if( i < eIdx ) {
        float result = 0.f;
        result += sharedWeights[0]*list[threadIdx.x];
        result += sharedWeights[1]*list[threadIdx.x+1];
        result += sharedWeights[2]*list[threadIdx.x+2];
        result += sharedWeights[3]*list[threadIdx.x+3];
        result += sharedWeights[4]*list[threadIdx.x+4];
        result += sharedWeights[5]*list[threadIdx.x+5];
        result += sharedWeights[6]*list[threadIdx.x+6];
        result /=7.f;
        out[i] = result;
    }
}

int main(int argc, char* argv[]) {
  if(argc!=2){
        printf("Usage %s N\n",argv[0]);
	return 1;
  }
  int N=atoi(argv[1]);
  FILE *fp = fopen("problem1.inp","r");
  int size = N * sizeof(float); 
  int wsize = (2 * RADIUS + 1) * sizeof(float); 
  //allocate resources
  float *weights, *in, *cuda_out; 
  hipHostMalloc((void **)&weights, wsize, hipHostMallocDefault);
  hipHostMalloc((void **)&in, size, hipHostMallocDefault); 
  hipHostMalloc((void **)&cuda_out, size, hipHostMallocDefault); 

  float *out     = (float *)malloc(size); 
  float time = 0.f;
  initializeWeights(weights, RADIUS);
  initializeArray(fp,in, N);
  
  //start inclusive timing
  hipEvent_t startIn,stopIn;
  hipEventCreate(&startIn);
  hipEventCreate(&stopIn);
  hipEventRecord(startIn, 0);
  
  float *d_weights;  hipMalloc(&d_weights, wsize);
  float *d_in;       hipMalloc(&d_in, size);
  float *d_out;      hipMalloc(&d_out, size);
  
  hipMemcpy(d_weights,weights,wsize,hipMemcpyHostToDevice);
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
  applyStencil1D<<<(N+511)/512, 512>>>(RADIUS, N-RADIUS, d_weights, d_in, d_out);
  hipMemcpy(cuda_out, d_out, size, hipMemcpyDeviceToHost);

  //stop inclusive timing
  hipEventRecord(stopIn, 0);     
  hipEventSynchronize(stopIn);
  hipEventElapsedTime(&time, startIn, stopIn);     
  hipEventDestroy(startIn); 
  hipEventDestroy(stopIn);

  applyStencil1D_SEQ(RADIUS, N-RADIUS, weights, in, out);
  int nDiffs = checkResults(RADIUS, N-RADIUS, cuda_out, out);
  if(nDiffs)printf("Test Failed\n"); // This should never print
  printf("%f\n%f\n",cuda_out[N-RADIUS-1],time);
  //free resources 
  hipHostFree(weights); hipHostFree(in); hipHostFree(cuda_out); 
  free(out);
  hipFree(d_weights);  hipFree(d_in);  hipFree(d_out);
  return 0;
}
