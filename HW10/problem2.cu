#ifdef _WIN32
#  define NOMINMAX 
#endif

// includes, system

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <assert.h>

#define BLOCK_SIZE 1024

__global__ void reduction(float *g_data, float *d_out, int n)
{
	int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	//int bSize = blockDim.x;
    int startSize = BLOCK_SIZE;
    	
	if(threadId >= n) {
		g_data[blockId*BLOCK_SIZE + threadId] = 0.0;
	}
    //synchronize the threads to make sure all the data is loaded
    __syncthreads();
    
    for(unsigned int i=startSize/2;i>0;i>>=1){
        //add the second half of the data to the first half
        if(threadId < i){
            g_data[blockId*BLOCK_SIZE + threadId] += g_data[blockId*BLOCK_SIZE + threadId+i];
        }
        //synchronize the threads to make sure all the caculation is made
        __syncthreads();
    }
    //copy the result back
    if(threadId==0){
		d_out[blockId] = g_data[blockId*BLOCK_SIZE];
    }
}

float reductionOnDevice(float *d_in, int num) {
	int blockx = (num + BLOCK_SIZE - 1)/BLOCK_SIZE;
	dim3 dimGrid(blockx, 1, 1);
	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	float *d_out;
	assert(hipSuccess == hipMalloc(&d_out, blockx*sizeof(float)));
	float *dd_out;
	float *ddd_out;
	float *ret = (float *)malloc(sizeof(float));
	
	int t = (blockx == 1) ? num:BLOCK_SIZE;
	reduction<<<dimGrid, dimBlock>>>(d_in, d_out, t);
	if(blockx == 1) {
		hipMemcpy(ret, d_out, sizeof(float), hipMemcpyDeviceToHost);
		return ret[0];
	}
	
	float *k = (float *)malloc(5*sizeof(float));
	
	hipMemcpy(k, d_out, 2*sizeof(float), hipMemcpyDeviceToHost);
	
	if(blockx > BLOCK_SIZE){
		//use several blocks in second level reduction
		int blockxx = (blockx + BLOCK_SIZE -1)/BLOCK_SIZE;
		dim3 dimGridd(blockxx, 1, 1);
		int tt;
		tt = (blockx == 1) ? num:BLOCK_SIZE;
		hipMalloc(&dd_out, blockxx*sizeof(float));
		reduction<<<dimGridd, dimBlock>>>(d_out,dd_out,tt);
		//can use a single block
		hipMalloc(&ddd_out, sizeof(float));
		reduction<<<1, dimBlock>>>(dd_out,ddd_out,blockxx);
		hipMemcpy(ret, ddd_out, sizeof(float), hipMemcpyDeviceToHost);
		hipFree(dd_out);
		hipFree(ddd_out);
	} else {
		//can use a single block
		hipMalloc(&ddd_out, sizeof(float));
		reduction<<<1, dimBlock>>>(d_out,ddd_out,blockx);
		hipMemcpy(ret, ddd_out, sizeof(float), hipMemcpyDeviceToHost);
		hipFree(ddd_out);
	}
	hipFree(d_out);
	return ret[0];
}

float* read_array(const char* filename, int len) {
	float *x = (float*) malloc(len * sizeof(float));
	FILE *fp = fopen(filename, "r");
        for( int i=0; i<len; i++){
		int r=fscanf(fp,"%f",&x[i]);		
		if(r == EOF){
			rewind(fp);
		}
		x[i]-=5;
         }
	fclose(fp);
	return x;
}
/*
float* read_array(const char* filename, int len) {
	float *x = (float*) malloc(len * sizeof(float));
	FILE *fp = fopen(filename, "r");
	for (int i = 0; i < len; i++) {
		fscanf(fp, "%f", &x[i]);
	}
	fclose(fp);
	return x;
}
*/
void computeSum( float* reference, float* idata, const unsigned int len) 
{
  reference[0] = 0;
  double total_sum = 0;
  unsigned int i;
  for( i = 0; i < len; ++i) 
  {
      total_sum += idata[i];
  }
  *reference = total_sum;
}

int main( int argc, char** argv) 
{
	if(argc != 2) {
		fprintf(stderr, "usage: ./problem2 N\n");
		exit(1);
	}
	int num_elements = atoi(argv[1]);

	float* h_data=read_array("problem1.inp",num_elements);
	
	float reference = 1.0f;  
	computeSum(&reference , h_data, num_elements);

	int size = num_elements*sizeof(float);
	float *d_in;
	assert(hipSuccess == hipMalloc((void**)&d_in, size));

	//start inclusive timing
	float time;
	hipEvent_t startIn,stopIn;
	hipEventCreate(&startIn);
	hipEventCreate(&stopIn);
	hipEventRecord(startIn, 0);	
	
	assert(hipSuccess == hipMemcpy(d_in, h_data, size, hipMemcpyHostToDevice));
	//float result = computeOnDevice(h_data, num_elements);
	float result = reductionOnDevice(d_in, num_elements);
	
	//stop inclusive timing
	hipEventRecord(stopIn, 0);     
	hipEventSynchronize(stopIn);
	hipEventElapsedTime(&time, startIn, stopIn);     
	hipEventDestroy(startIn); 
	hipEventDestroy(stopIn);

	// Run accuracy test
	float epsilon = 0.3f;
	unsigned int result_regtest = (abs(result - reference) <= epsilon);

	if(!result_regtest)	printf("Test failed device: %f  host: %f\n",result,reference);
	//print the outputs
	printf("%d\n%f\n%f\n",num_elements, result, time);
	//printf("%f\n", time);
	// cleanup memory
	hipFree(d_in);  
	//cudaFree(d_out);
	free( h_data);
	return 0;
}
