#include<iostream>
#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void sumKernel(double* dA, double* dB, double* dC, int size)
{
    int index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index < size){
        dC[index] = dA[index] + dB[index];
    }
}


int main( int argc, char *argv[])
{

	if(argc!=3)
	{
		printf("Invalid argument Usage: ./problem3 N M");
		return 0;
	}

	FILE *fpA,*fpB;
	int N = atoi(argv[1]);
	int M = atoi(argv[2]);
	int threadSize = M;
	int blockSize = N/threadSize + 1;
	double *hA= new double[N];
	double *hB= new double[N];
	double *hC=  new double[N];
	double *refC=  new double[N]; // Used to verify functional correctness
	double *dA,*dB,*dC;  // You may use these to allocate memory on gpu
	//defining variables for timing
	hipEvent_t startEvent_inc, stopEvent_inc, startEvent_exc, stopEvent_exc;
	hipEventCreate(&startEvent_inc);
	hipEventCreate(&stopEvent_inc);
	hipEventCreate(&startEvent_exc);
	hipEventCreate(&stopEvent_exc);
	float elapsedTime_inc, elapsedTime_exc;

	//reading files
	fpA = fopen("inputA.inp", "r");
	fpB= fopen("inputB.inp", "r");


	for (int i=0;i<N;i++){    
		fscanf(fpA, "%lf",&hA[i]);
	}
	for (int i=0;i<N;i++){
		fscanf(fpB, "%lf",&hB[i]);
	}

      for(int i=0;i<N;i++)
        refC[i]=hA[i]+hB[i];

    hipMalloc((void**) &dA,sizeof(double)*N);
    hipMalloc((void**) &dB,sizeof(double)*N);
    hipMalloc((void**) &dC,sizeof(double)*N);
	hipEventRecord(startEvent_inc,0); // starting timing for inclusive
	// TODO allocate memory for arrays and copay array A and B
	//copy the hA into dA, hB into dB
    hipMemcpy(dA, hA, sizeof(double)*N, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(double)*N, hipMemcpyHostToDevice);

	hipEventRecord(startEvent_exc,0); // staring timing for exclusive
	//launch the kernel to compute the sum
    sumKernel<<<blockSize, threadSize>>>(dA,dB,dC,N);

	hipEventRecord(stopEvent_exc,0);  // ending timing for exclusive
	hipEventSynchronize(stopEvent_exc);   
	hipEventElapsedTime(&elapsedTime_exc, startEvent_exc, stopEvent_exc);
	
	// TODO copy data back
	hipMemcpy(hC, dC, sizeof(double)*N, hipMemcpyDeviceToHost);

	hipEventRecord(stopEvent_inc,0);  //ending timing for inclusive
	hipEventSynchronize(stopEvent_inc);   
	hipEventElapsedTime(&elapsedTime_inc, startEvent_inc, stopEvent_inc);


	//verification
	int count=0;
	for(int i=0;i<N;i++)
	{
		if(hC[i]!=refC[i])
		{
			count++;
		}
	}
	if(count!=0) // This should never be printed in correct code
		std::cout<<"Error at "<< count<<" locations\n";
	std::cout<<N<<"\n"<<M<<"\n"<<elapsedTime_exc<<"\n"<<elapsedTime_inc<<"\n"<<hC[N-1]<<"\n";
	//std::cout<<elapsedTime_exc<<"\n"<<elapsedTime_inc<<"\n";
	//freeing memory
	delete[] hA,hB,hC,refC;     

	// TODO free CUDA memory allocated
	if(dA)  hipFree(dA);
    if(dB)  hipFree(dB);
    if(dC)  hipFree(dC);
    hipEventDestroy(startEvent_inc);
    hipEventDestroy(stopEvent_inc);
    hipEventDestroy(startEvent_exc);
    hipEventDestroy(stopEvent_exc);

	return 0;
}
