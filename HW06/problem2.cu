#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>

//A kernel calculates the sum of the threadidx and blockidx
__global__ void sumKernel(int* data, int size)
{
    int threadi = threadIdx.x;
    int blocki = blockIdx.x;
    int value = blocki*blockDim.x + threadi;
    if(value < size){
        data[value] = threadi+blocki;
    }

}


int main(int argc, char* argv[])
{
    const int totalSize = 16;
    const int blockSize = 2;
    const int threadSize  = 8;

    int hostArr[totalSize];

    //allocate the memory
    int *dArray;
    hipMalloc((void **)&dArray,sizeof(int)*totalSize);

    sumKernel<<<blockSize, threadSize>>>(dArray, totalSize);
    
    hipMemcpy(&hostArr, dArray, sizeof(int)*totalSize, hipMemcpyDeviceToHost); 

    //print the output
    int i;
    for(i=0; i<totalSize;i++){
        printf("%d\n", hostArr[i]);
    }

    //clean up the memory
    hipFree(dArray);
    return 0;
}
