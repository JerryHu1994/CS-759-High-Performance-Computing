/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifndef _VECTOR_REDUCTION_KERNEL_H_
#define _VECTOR_REDUCTION_KERNEL_H_

#define NUM_ELEMENTS 1024


// **===----------------- Modify this function ---------------------===**
//! @param g_idata  input data in global memory
//                  result is expected in index 0 of g_idata
//! @param n        input number of elements to scan from input data
// **===------------------------------------------------------------------===**
__global__ void reduction(float *g_data, int n)
{
	int threadId = threadIdx.x;
	
	int index = threadId;
    int startSize = n/2;
    __shared__ double s_data[512];
    
    //copy the data into the shared memory
    if(index < n/2){
        s_data[threadId] = g_data[index*2] + g_data[index*2+1];
    }else{
        s_data[threadId] = 0;
    }
    //synchronize the threads to make sure all the data is loaded
    __syncthreads();
    
    for(unsigned int i=startSize/2;i>0;i>>=1){
        //add the second half of the data to the first half
        if(threadId < i){
            s_data[threadId] += s_data[threadId+i];
        }
        //synchronize the threads to make sure all the caculation is made
        __syncthreads();
		
    }
    //copy the result back
    if(threadId==0){
        g_data[threadId] = s_data[0];
    }
}

#endif // #ifndef _VECTOR_REDUCTION_KERNEL_H_
